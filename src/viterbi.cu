
#include <hip/hip_runtime.h>
#include <iostream>
#include <iterator>
#include <stdio.h>
#include <vector>
#include <algorithm>
#include <ctime>

#define NUM_STATES 64
#define NUM_INPUT_SYMBOLS 2
#define NUM_OUTPUT_SYMBOLS 4
#define NUM_BLOCKS 1024
#define TRACEBACK 64

__constant__ const char outputs[NUM_STATES][NUM_INPUT_SYMBOLS] = { { 0, 3 }, {
		3, 0 }, { 1, 2 }, { 2, 1 }, { 0, 3 }, { 3, 0 }, { 1, 2 }, { 2, 1 }, { 3,
		0 }, { 0, 3 }, { 2, 1 }, { 1, 2 }, { 3, 0 }, { 0, 3 }, { 2, 1 },
		{ 1, 2 }, { 3, 0 }, { 0, 3 }, { 2, 1 }, { 1, 2 }, { 3, 0 }, { 0, 3 }, {
				2, 1 }, { 1, 2 }, { 0, 3 }, { 3, 0 }, { 1, 2 }, { 2, 1 },
		{ 0, 3 }, { 3, 0 }, { 1, 2 }, { 2, 1 }, { 2, 1 }, { 1, 2 }, { 3, 0 }, {
				0, 3 }, { 2, 1 }, { 1, 2 }, { 3, 0 }, { 0, 3 }, { 1, 2 },
		{ 2, 1 }, { 0, 3 }, { 3, 0 }, { 1, 2 }, { 2, 1 }, { 0, 3 }, { 3, 0 }, {
				1, 2 }, { 2, 1 }, { 0, 3 }, { 3, 0 }, { 1, 2 }, { 2, 1 },
		{ 0, 3 }, { 3, 0 }, { 2, 1 }, { 1, 2 }, { 3, 0 }, { 0, 3 }, { 2, 1 }, {
				1, 2 }, { 3, 0 }, { 0, 3 }, };

typedef struct {
	char w;
} bm_t;

typedef struct {
	char w;
	char prev;
} pm_t;

__global__ void calc_bm(char *encData, char *decData,
		pm_t (*pm)[NUM_BLOCKS][TRACEBACK][NUM_STATES]) {
	int state = threadIdx.x;
	int block = blockIdx.x;
	int offset = block * TRACEBACK;

	for (int tms = 1; tms < TRACEBACK; tms++) {

		int _tms = tms + offset;

		int prevState = (state & (~32)) << 1;
		int prevSymbol = (state >> 5) & 1;

		char data = encData[_tms];
		char d0 = data ^ outputs[prevState][prevSymbol];
		char d1 = data ^ outputs[prevState + 1][prevSymbol];

		char c0 = ((d0 >> 1) & 1) + (d0 & 1);
		char c1 = ((d1 >> 1) & 1) + (d1 & 1);

		char w0 = c0 + (*pm)[block][tms][prevState].w;
		char w1 = c1 + (*pm)[block][tms][prevState + 1].w;

		int idx = ((tms + 1) & (TRACEBACK - 1));

//		(*pm)[block][idx][state].w = w0 < w1 ? w0 : w1;
//		(*pm)[block][idx][state].symbol = prevSymbol;
//		(*pm)[block][idx][state].prev = w0 < w1 ? prevState : prevState + 1;

		(*pm)[block][idx][state] = {
			.w = w0 < w1 ? w0 : w1,
			.prev = w0 < w1 ? char(prevState) : char(prevState + 1)
		};

		__syncthreads();
	}

	if (state == 0) {
		char min = 120;
		int minIdx = 0;
		int tms = 0;
		for (int i = 0; i < NUM_STATES; i++) {
			char w = (*pm)[block][tms][i].w;
			if (min > w) {
				min = w;
				minIdx = i;
			}
		}

		int count = 0;
		int _tms = TRACEBACK - 1;
		while (count < TRACEBACK) {
			decData[_tms] = (minIdx & 32) >> 5;
			minIdx = (*pm)[block][tms][minIdx].prev;
			tms--;
			while (tms < 0) {
				tms += TRACEBACK;
			}
			_tms = tms + offset - 1;
			count++;
		}
	}

}

std::vector<int> traceback(pm_t (*pm)[NUM_BLOCKS][TRACEBACK][NUM_STATES],
		unsigned long tm) {

	int min = 100000;
	int minIdx = -1;

	int tms = tm % TRACEBACK;

	std::vector<int> tb;
	for (int i = 0; i < NUM_STATES; i++) {
		if (min > (*pm)[0][tms][i].w) {
			min = (*pm)[0][tms][i].w;
			minIdx = i;
		}
	}

	int count = 0;

	while (count < TRACEBACK) {
		tb.insert(tb.begin(), /*(*pm)[0][tms][minIdx].symbol*/
		(minIdx & 32) >> 5);

		minIdx = (*pm)[0][tms][minIdx].prev;
		tms--;
		while (tms < 0) {
			tms += TRACEBACK;
		}
		count++;
	}

	return tb;
}

// rng(2)
const char data[128] = { 0, 0, 0, 0, 1, 1, 1, 0, 1, 1, 1, 1, 0, 0, 1, 0, 0, 1,
		1, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 1, 0, 1, 0, 0, 1, 0, 1, 0, 1, 0, 0, 1,
		1, 1, 0, 0, 1, 0, 1, 1, 1, 0, 0, 1, 1, 1, 1, 1, 0, 0, 0, 1, 1, 1, 1, 1,
		1, 0, 0, 0, 0, 1, 0, 0, 1, 1, 1, 0, 1, 0, 0, 1, 1, 1, 1, 1, 0, 1, 1, 0,
		1, 1, 0, 1, 1, 0, 1, 0, 1, 1, 1, 0, 1, 0, 0, 1, 1, 0, 0, 0, 0, 1, 0, 0,
		0, 0, 1, 0, 1, 0, 1, 0, 0, 1, 0, 1, 1, 1 };

// rng(3)
//const char data[128] = { 1, 1, 0, 1, 0, 1, 1, 1, 1, 0, 1, 1, 0, 0, 1, 1, 1, 0,
//		1, 0, 1, 0, 1, 1, 1, 1, 1, 0, 0, 0, 1, 0, 0, 1, 1, 0, 1, 0, 1, 0, 0, 1,
//		0, 0, 1, 1, 0, 0, 0, 0, 0, 1, 0, 0, 1, 1, 1, 1, 1, 0, 1, 1, 0, 1, 0, 1,
//		0, 1, 1, 1, 1, 0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 0, 1, 1, 0, 1, 1, 0, 0,
//		1, 1, 0, 1, 0, 1, 0, 0, 1, 1, 0, 1, 1, 0, 0, 0, 1, 0, 1, 1, 1, 1, 1, 1,
//		0, 0, 0, 1, 0, 1, 1, 0, 1, 0, 1, 1, 1, 0 };

int main(int argc, char **argv) {

	pm_t (*pm)[NUM_BLOCKS][TRACEBACK][NUM_STATES];
	char *encData;
	char *decData;
	hipMallocManaged(&pm, NUM_BLOCKS * TRACEBACK * NUM_STATES * sizeof(pm_t));
	hipMallocManaged(&encData, TRACEBACK * NUM_BLOCKS);
	hipMallocManaged(&decData, TRACEBACK * NUM_BLOCKS);

	for (int i = 0; i < TRACEBACK * NUM_BLOCKS; i++) {
		int d = data[(i % 64) * 2] << 1 | data[(i % 64) * 2 + 1];
		encData[i] = d;
	}

	int count = 1000;
	clock_t begin = clock();

	for (int q = 0; q < count; q++) {
		calc_bm<<<dim3(NUM_BLOCKS), dim3(NUM_STATES)>>>(encData, decData, pm);
		hipDeviceSynchronize();

	}

	clock_t end = clock();
	for (int j = 0; j < 2; j++) {
		for (int i = 0; i < 64; i++)
			std::cout << int(decData[i]) << ", ";
		std::cout << std::endl;
	}

	std::cout << std::endl;
	std::cout << "done "
			<< ((count * TRACEBACK * NUM_BLOCKS * 2)
					/ (double(end - begin) / CLOCKS_PER_SEC)) << " " << " "
			<< (double(end - begin) / CLOCKS_PER_SEC) << std::endl;

	hipFree(pm);
	hipFree(encData);
	hipFree(decData);

	return 0;
}
